
#include <hip/hip_runtime.h>
#include<stdio.h>

//GPU CODE!
__global__ void add(int *a, int *b, int *c){
  *c = *a + *b;
}
//CPU CODE
int main(void){
  int a, b, c; //host variables
  int *d_a, *d_b, *d_c; //GPU copies of host variables

  a = 9;
  b = 32;

  int size = sizeof(int);

//Allocate space from GPU for host copies
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

//copy input to GPU
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

//GPU kernel launcher
  add<<<1,1>>>(d_a, d_b, d_c);

  //Copy result from GPU
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  printf("%d\n",c);

  //cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
