#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER               float
#define precision            "float"
#define Kernel_cycles        10000
#define Cycles               512
#define Cycles2              512
#define BLOCKS               8
#define THREADS              128
#define Correct_value        187.5

__global__ void gpu_monte_carlo(NUMBER *estimate, hiprandState *states, int seed){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  NUMBER ans, ans2 = 0;
  NUMBER x, y, a, b, c, d;
  a = c = 0;
  b = d = 5;
  hiprand_init(seed, tid, 0, &states[tid]);

  for (long i = 0; i < Cycles2; i++){
    ans = 0;
    for (long i = 0; i < Cycles; ++i){
      x = (a+(b-a))*hiprand_uniform(&states[tid]);
      y = (c+(d-c))*hiprand_uniform(&states[tid]);
      ans += (2*x+y);
    }
    ans2 += ans/(NUMBER)Cycles;
  }
  estimate[tid] = ans2*(((b-a)*(d-c)) / (NUMBER) Cycles2) ;
}


int main(){
  clock_t t[2];
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  //variables and pointers
  NUMBER host[BLOCKS * THREADS];
  NUMBER *dev;
  hiprandState *devStates;
  int seed = 1;
  double integral_gpu;
  printf("# of cycles per kernel = %d, # of blocks = %d, # of threads/block = %d.\n",
  Kernel_cycles, BLOCKS, THREADS);
  hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(NUMBER));
  hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );
  t[0] = clock();

  for (long i = 0; i < Kernel_cycles; i++) {
    seed +=1;
    gpu_monte_carlo<<<BLOCKS, THREADS,1,stream1>>>(dev, devStates, seed);
    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(NUMBER), hipMemcpyDeviceToHost);
    for (long i = 0; i < BLOCKS * THREADS; ++i){
      integral_gpu += host[i];
    }

  }
  t[1] = clock();
  integral_gpu /= (BLOCKS * THREADS * (long)Kernel_cycles);
  printf("Precision %s\n",precision);
  printf("GPU calculation  %f  s.\n", ((t[1]-t[0]))/(double)CLOCKS_PER_SEC);
  printf("CUDA estimate of 2*x+y = %f [error of %f]\n", integral_gpu, integral_gpu - Correct_value);

  hipFree(dev);
  hipFree(devStates);
  return 0;
}

