
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define Trials               1000
#define BLOCKS               512
#define THREADS              512
#define Correct_value        187.5


__global__ void gpu_monte_carlo(double *estimate, hiprandState *states)
{
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double ans = 0;
    double x, y, a, b, c, d;
    a = c = 0;
    b = d = 5;
    hiprand_init(1234, tid, 0, &states[tid]);

    for (int i = 0; i < Trials; ++i)
    {
        x = (a+(b-a))*hiprand_uniform(&states[tid]);
        y = (c+(d-c))*hiprand_uniform(&states[tid]);
        ans += (2*x+y);
    }
    estimate[tid] = ans*(((b-a)*(d-c)) / (double) Trials) ;
}


int main ()
{
    clock_t start, stop;
    double host[BLOCKS * THREADS];
    double *dev;
    hiprandState *devStates;
    printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n",
    Trials, BLOCKS, THREADS);
    start = clock();
    hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(double));
    hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );
    gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);
    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(double), hipMemcpyDeviceToHost);
    double integral_gpu;
    for (int i = 0; i < BLOCKS * THREADS; ++i)
    {
        integral_gpu += host[i];
    }

    integral_gpu /= (BLOCKS * THREADS);
    stop = clock();
    printf("GPU 2*x+y calculated in %f s.\n", (stop-start)/(double)CLOCKS_PER_SEC);
    printf("CUDA estimate of 2*x+y = %f [error of %f]\n", integral_gpu, integral_gpu - Correct_value);

    return 0;
}
