#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define NUMBER               float
#define precision            "float"
#define Cycles               1000
#define Cycles2              512
#define BLOCKS               8
#define THREADS              128
#define Correct_value        187.5

__global__ void gpu_monte_carlo(NUMBER *estimate, hiprandState *states){
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  NUMBER ans, ans2 = 0;
  NUMBER x, y, a, b, c, d;
  a = c = 0;
  b = d = 5;
  hiprand_init(1234, tid, 0, &states[tid]);

  for (long i = 0; i < Cycles2; i++){
    ans = 0;
    for (long i = 0; i < Cycles; ++i){
      x = (a+(b-a))*hiprand_uniform(&states[tid]);
      y = (c+(d-c))*hiprand_uniform(&states[tid]);
      ans += (2*x+y);
    }
    ans2 += ans/(NUMBER)Cycles;
  }
  estimate[tid] = ans2*(((b-a)*(d-c)) / (NUMBER) Cycles2) ;
}

int main (){
  clock_t t[6];
  NUMBER host[BLOCKS * THREADS];
  NUMBER *dev;
  hiprandState *devStates;
  printf("# of cycles per thread = %d, # of blocks = %d, # of threads/block = %d.\n",
  Cycles, BLOCKS, THREADS);
  t[0] = clock();
  hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(NUMBER));
  t[1] = clock();
  hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );
  t[2] = clock();
  gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates);
  t[3] = clock();
  hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(NUMBER), hipMemcpyDeviceToHost);
  t[4] = clock();
  NUMBER integral_gpu;
  for (long i = 0; i < BLOCKS * THREADS; ++i){
    integral_gpu += host[i];
  }
  t[5] = clock();
  integral_gpu /= (NUMBER)(BLOCKS * THREADS);
  printf("Precision %s\n",precision);
  printf("CPU mem allocation  %f  ms.\n", ((t[1]-t[0])*1000.0)/(double)CLOCKS_PER_SEC);
  printf("GPU mem allocation  %f  ms.\n", ((t[2]-t[1])*1000.0)/(double)CLOCKS_PER_SEC);
  printf("GPU Kernel launch  %f  ms.\n", ((t[3]-t[2])*1000.0)/(double)CLOCKS_PER_SEC);
  printf("GPU calculation + copy from GPU mem to CPU mem  %f  ms.\n", ((t[4]-t[3])*1000.0)/(double)CLOCKS_PER_SEC);
  printf("CPU list loop  %f  ms.\n", ((t[4]-t[3])*1000.0)/(double)CLOCKS_PER_SEC);
  printf("CUDA estimate of 2*x+y = %f [error of %f]\n", integral_gpu, integral_gpu - Correct_value);
  return 0;
}
