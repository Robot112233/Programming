#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

#define LOOP 10000000
#define N 1
#define X_max 5.0
#define X_min 0.0
#define Y_max 5.0
#define Y_min 0.0
//GPU CODE!
__global__ void add(double *C_inputA, double *C_inputB, double *C_output){
  for (int i = 0; i < LOOP; i++) {
    C_output[0] = C_output[0]+2*C_inputA[blockIdx.x]+C_inputB[blockIdx.x];
  }
}

void var_A(double* C_inputA){
  for (int i = 0; i < LOOP; ++i){
    double d = rand()/(double)RAND_MAX;
    C_inputA[i] = X_min+(X_max-X_min)*d;
  }
}
void var_B(double* C_inputB){
  for (int i = 0; i < LOOP; ++i){
    double d = rand()/(double)RAND_MAX;
    C_inputB[i] = Y_min+(Y_max-Y_min)*d;
  }
}

//CPU CODE
int main(void){
  srand (time(0));

  double *C_inputA, *C_inputB, *C_output; //host copies of variables
  double *G_inputA, *G_inputB, *G_output; //GPU copies of host variables
  double size = LOOP*N * sizeof(double);

//Allocate space from GPU for host copies
  hipMalloc((void **) &G_inputA, size);
  hipMalloc((void **) &G_inputB, size);
  hipMalloc((void **) &G_output, size);

//Allocate space for cpu copies
  C_inputA = (double *)malloc(size); var_A(C_inputA);
  C_inputB = (double *)malloc(size); var_B(C_inputB);
  C_output = (double *)malloc(size);

//copy input to GPU
  hipMemcpy(G_inputA, C_inputA, size, hipMemcpyHostToDevice);
  hipMemcpy(G_inputB, C_inputB, size, hipMemcpyHostToDevice);

//GPU kernel launcher
  add<<<N,1>>>(G_inputA, G_inputB, G_output);

  //Copy result from GPU
  hipMemcpy(C_output, G_output, size, hipMemcpyDeviceToHost);
  double ans = *C_output;
  ans = (((X_max-X_min)*(Y_max-Y_min))/LOOP)*ans;
  printf("%e\n",ans);
  //cleanup
  free(C_inputA);
  free(C_inputB);
  free(C_output);
  hipFree(G_inputA);
  hipFree(G_inputB);
  hipFree(G_output);

  return 0;
}
